#include "sl_device.cuh"

extern vector<string> strLITERAL;
extern vector<double> nbrLITERAL;

double* d_gmem = NULL;

void sl_run_device(int devId, int maxProc, int nBlocks, int nThreads, double* host_List)
{
	hipError_t error = hipSetDevice(devId);

	if (error != hipSuccess)
	{
		 printf("hipSetDevice returned error code %d, line(%d)\n", error, __LINE__);
		 exit(EXIT_FAILURE);
	}


     // get number of SMs on this GPU
     error = hipGetDevice(&devId);

     if (error != hipSuccess)
     {
         printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
         exit(EXIT_FAILURE);
     }

    int IndexSiz, CodeArrSiz, DmemSiz, GmemSiz, nbrSiz;
    int GSiz, LSiz, spReg;

    int *h_Index, *d_Index;
    int *h_CodeArr, *d_CodeArr;
    d_SymTbl *h_GTbl, *h_LTbl, *d_GTbl, *d_LTbl;
    double *h_Dmem, *h_Gmem, *h_nbrLITERAL, *d_Dmem, *d_Gmem, *d_nbrLITERAL;

    RUN_PARM *d_runParm;

	Stack* d_stk;
	double* d_stack;
	TokenSet* d_code;

    IndexSiz = (int)Ind.size();
    CodeArrSiz = (int)intercode.size();

    h_Index = new int[IndexSiz];
    h_CodeArr = new int[CodeArrSiz];

    for (int i=0; i< IndexSiz; i++) h_Index[i]=Ind[i];

    for (int i=0; i< CodeArrSiz; i++) {
        h_CodeArr[i] = intercode[i];
    }

    if (hipSuccess != hipMalloc((void **) &d_Index, sizeof(int)*IndexSiz))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
    hipMemcpy(d_Index, h_Index, sizeof(int)*IndexSiz, hipMemcpyHostToDevice);

    if (hipSuccess != hipMalloc((void **) &d_CodeArr, sizeof(int)*CodeArrSiz))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
    hipMemcpy(d_CodeArr, h_CodeArr, sizeof(int)*CodeArrSiz, hipMemcpyHostToDevice);

    spReg = Dmem.size();

    int BigFram = 0, SizGTbl = 0;
    SizGTbl = Gtable.size();

    for (int i=0; i < SizGTbl; i++) {
        int Fram = Gtable[i].frame;
        if (BigFram < Fram) BigFram = Fram;
    }

    Dmem.resize(spReg + BigFram);

    GSiz = Gtable.size();
    LSiz = Ltable.size();

    h_GTbl = new d_SymTbl[GSiz];
    h_LTbl = new d_SymTbl[LSiz];
    for (int i=0; i<GSiz; i++) {
		h_GTbl[i].nmKind =Gtable[i].nmKind;
		h_GTbl[i].dtTyp	=Gtable[i].dtTyp;
		h_GTbl[i].aryLen =Gtable[i].aryLen;
		h_GTbl[i].args =Gtable[i].args;
		h_GTbl[i].adrs =Gtable[i].adrs;
		h_GTbl[i].frame =Gtable[i].frame;
	}

    for (int i=0; i<LSiz; i++) {
		h_LTbl[i].nmKind =Ltable[i].nmKind;
		h_LTbl[i].dtTyp	=Ltable[i].dtTyp;
		h_LTbl[i].aryLen =Ltable[i].aryLen;
		h_LTbl[i].args =Ltable[i].args;
		h_LTbl[i].adrs =Ltable[i].adrs;
		h_LTbl[i].frame =Ltable[i].frame;
	}

    if (hipSuccess != hipMalloc((void **) &d_GTbl, sizeof(d_SymTbl)*GSiz))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
    hipMemcpy(d_GTbl, h_GTbl, sizeof(d_SymTbl)*GSiz, hipMemcpyHostToDevice);

    if (hipSuccess != hipMalloc((void **) &d_LTbl, sizeof(d_SymTbl)*LSiz))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
    hipMemcpy(d_LTbl, h_LTbl, sizeof(d_SymTbl)*LSiz, hipMemcpyHostToDevice);

    if (hipSuccess != hipMalloc((void **) &d_runParm, sizeof(RUN_PARM)*maxProc))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }

    DmemSiz = Dmem.size();
    GmemSiz = Gmem.size();
    nbrSiz = nbrLITERAL.size();

    h_Dmem = new double[DmemSiz * maxProc];
    h_Gmem = new double[GmemSiz];
    h_nbrLITERAL = new double[nbrSiz+10];

    for (int i=0; i< DmemSiz * maxProc; i++) {
		int lo_i = i % DmemSiz;
		h_Dmem[i] = Dmem.get(lo_i);
	}

    for (int i=0; i< GmemSiz; i++) h_Gmem[i] = Gmem.get(i);
    for (int i=0; i< nbrSiz; i++)  h_nbrLITERAL[i] = nbrLITERAL[i];

    if (hipSuccess != hipMalloc(&d_Dmem, sizeof(double)*DmemSiz * maxProc))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
    hipMemcpy(d_Dmem, h_Dmem, sizeof(double)*DmemSiz * maxProc, hipMemcpyHostToDevice);

    if (hipSuccess != hipMalloc(&d_Gmem, sizeof(double)*GmemSiz))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
    hipMemcpy(d_Gmem, h_Gmem, sizeof(double)*GmemSiz, hipMemcpyHostToDevice);

    if (hipSuccess != hipMalloc(&d_nbrLITERAL, sizeof(double)*nbrSiz))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
    hipMemcpy(d_nbrLITERAL, h_nbrLITERAL, sizeof(double)*nbrSiz, hipMemcpyHostToDevice);

	if (hipSuccess != hipMalloc((void **) &d_stk, sizeof(Stack) * maxProc))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
	if (hipSuccess != hipMalloc((void **) &d_stack, sizeof(double) * MAXSIZE_ * maxProc))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
	if (hipSuccess != hipMalloc((void **) &d_code, sizeof(TokenSet) * 2 * maxProc))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }


	int maxLoop = maxProc / (nThreads * nBlocks);

	for (int nloop = 0; nloop < maxLoop + 1; nloop ++)
	{
    	sl_Exe_global<<<nBlocks, nThreads>>>(nloop, nBlocks, nThreads, maxProc, DmemSiz, IndexSiz, spReg,
								d_runParm, d_stk, d_GTbl, d_LTbl, d_Index, d_CodeArr, d_Dmem,
								d_Gmem, d_nbrLITERAL, d_code, d_stack);
	}

    hipMemcpy(h_Dmem, d_Dmem, sizeof(double)*DmemSiz * maxProc, hipMemcpyDeviceToHost);
    hipMemcpy(h_Gmem, d_Gmem, sizeof(double)*GmemSiz, hipMemcpyDeviceToHost);

		for (int i=0; i<GmemSiz; i++)
		{
				host_List[i] = h_Gmem[i];
		}

	delete [] h_Index;
	hipFree(d_Index);
	delete [] h_CodeArr;
	hipFree(d_CodeArr);
	delete [] h_GTbl;
	delete [] h_LTbl;
	hipFree(d_GTbl);
	hipFree(d_LTbl);
	delete [] h_Dmem;
	delete [] h_Gmem;
	delete [] h_nbrLITERAL;
	hipFree(d_Dmem);
	hipFree(d_Gmem);
	hipFree(d_nbrLITERAL);

	hipFree(d_runParm);

	hipFree(d_stk);
	hipFree(d_stack);
	hipFree(d_code);
}

void sl_run_device_H5(int devId, int maxProc, int nBlocks, int nThreads)
{
	//hipSetDevice(devId);

    int IndexSiz, CodeArrSiz, DmemSiz, GmemSiz, nbrSiz;
    int GSiz, LSiz, spReg;

    int *h_Index, *d_Index;
    int *h_CodeArr, *d_CodeArr;
    d_SymTbl *h_GTbl, *h_LTbl, *d_GTbl, *d_LTbl;
    double *h_Dmem, *h_Gmem, *h_nbrLITERAL, *d_Dmem, *d_Gmem, *d_nbrLITERAL;

    RUN_PARM *d_runParm;

	Stack* d_stk;
	double* d_stack;
	TokenSet* d_code;

    IndexSiz = (int)Ind.size();
    CodeArrSiz = (int)intercode.size();

    h_Index = new int[IndexSiz];
    h_CodeArr = new int[CodeArrSiz];

    for (int i=0; i< IndexSiz; i++) h_Index[i]=Ind[i];

    for (int i=0; i< CodeArrSiz; i++) {
        h_CodeArr[i] = intercode[i];
    }

    //std::cout<<IndexSiz<<std::endl;
    //std::cout << hipMalloc( &d_Index, sizeof(int)*IndexSiz) << std::endl;
    
    if (hipSuccess != hipMalloc((void **) &d_Index, sizeof(int)*IndexSiz))
    {
        std::cout << "Memory Over 1" << std::endl;
        return ;
    }

    hipMemcpy(d_Index, h_Index, sizeof(int)*IndexSiz, hipMemcpyHostToDevice);

    if (hipSuccess != hipMalloc((void **) &d_CodeArr, sizeof(int)*CodeArrSiz))
    {
        std::cout << "Memory Over 2" << std::endl;
        return;
    }
    hipMemcpy(d_CodeArr, h_CodeArr, sizeof(int)*CodeArrSiz, hipMemcpyHostToDevice);

    spReg = Dmem.size();

    int BigFram = 0, SizGTbl = 0;
    SizGTbl = Gtable.size();

    for (int i=0; i < SizGTbl; i++) {
        int Fram = Gtable[i].frame;
        if (BigFram < Fram) BigFram = Fram;
    }

    Dmem.resize(spReg + BigFram);

    GSiz = Gtable.size();
    LSiz = Ltable.size();

    h_GTbl = new d_SymTbl[GSiz];
    h_LTbl = new d_SymTbl[LSiz];
    for (int i=0; i<GSiz; i++) {
		h_GTbl[i].nmKind =Gtable[i].nmKind;
		h_GTbl[i].dtTyp	=Gtable[i].dtTyp;
		h_GTbl[i].aryLen =Gtable[i].aryLen;
		h_GTbl[i].args =Gtable[i].args;
		h_GTbl[i].adrs =Gtable[i].adrs;
		h_GTbl[i].frame =Gtable[i].frame;
	}

    for (int i=0; i<LSiz; i++) {
		h_LTbl[i].nmKind =Ltable[i].nmKind;
		h_LTbl[i].dtTyp	=Ltable[i].dtTyp;
		h_LTbl[i].aryLen =Ltable[i].aryLen;
		h_LTbl[i].args =Ltable[i].args;
		h_LTbl[i].adrs =Ltable[i].adrs;
		h_LTbl[i].frame =Ltable[i].frame;
	}

    if (hipSuccess != hipMalloc((void **) &d_GTbl, sizeof(d_SymTbl)*GSiz))
    {
        std::cout << "Memory Over 3" << std::endl;
        return;
    }
    hipMemcpy(d_GTbl, h_GTbl, sizeof(d_SymTbl)*GSiz, hipMemcpyHostToDevice);

    if (hipSuccess != hipMalloc((void **) &d_LTbl, sizeof(d_SymTbl)*LSiz))
    {
        std::cout << "Memory Over 4" << std::endl;
        return;
    }
    hipMemcpy(d_LTbl, h_LTbl, sizeof(d_SymTbl)*LSiz, hipMemcpyHostToDevice);

    if (hipSuccess != hipMalloc((void **) &d_runParm, sizeof(RUN_PARM)*maxProc))
    {
        std::cout << "Memory Over 5" << std::endl;
        return;
    }

    DmemSiz = Dmem.size();
    GmemSiz = Gmem.size();
    nbrSiz = nbrLITERAL.size();

    h_Dmem = new double[DmemSiz * maxProc];
    //h_Gmem = new double[GmemSiz];
    h_nbrLITERAL = new double[nbrSiz+10];

    for (int i=0; i< DmemSiz * maxProc; i++) {
		int lo_i = i % DmemSiz;
		h_Dmem[i] = Dmem.get(lo_i);
	}

    //for (int i=0; i< GmemSiz; i++) h_Gmem[i] = Gmem.get(i);
    for (int i=0; i< nbrSiz; i++)  h_nbrLITERAL[i] = nbrLITERAL[i];

    if (hipSuccess != hipMalloc(&d_Dmem, sizeof(double)*DmemSiz * maxProc))
    {
        std::cout << "Memory Over 6" << std::endl;
        return;
    }
    hipMemcpy(d_Dmem, h_Dmem, sizeof(double)*DmemSiz * maxProc, hipMemcpyHostToDevice);
/*
    if (hipSuccess != hipMalloc(&d_Gmem, sizeof(double)*GmemSiz))
    {
        std::cout << "Memory Over 7" << std::endl;
        return;
    }
    hipMemcpy(d_Gmem, h_Gmem, sizeof(double)*GmemSiz, hipMemcpyHostToDevice);
*/
    if (hipSuccess != hipMalloc(&d_nbrLITERAL, sizeof(double)*nbrSiz))
    {
        std::cout << "Memory Over 8" << std::endl;
        return;
    }
    hipMemcpy(d_nbrLITERAL, h_nbrLITERAL, sizeof(double)*nbrSiz, hipMemcpyHostToDevice);

	if (hipSuccess != hipMalloc((void **) &d_stk, sizeof(Stack) * maxProc))
    {
        std::cout << "Memory Over 9" << std::endl;
        return;
    }
	if (hipSuccess != hipMalloc((void **) &d_stack, sizeof(double) * MAXSIZE_ * maxProc))
    {
        std::cout << "Memory Over 10" << std::endl;
        return;
    }
	if (hipSuccess != hipMalloc((void **) &d_code, sizeof(TokenSet) * 2 * maxProc))
    {
        std::cout << "Memory Over 11" << std::endl;
        return;
    }


	int maxLoop = maxProc / (nThreads * nBlocks);

	for (int nloop = 0; nloop < maxLoop + 1; nloop ++)
	{
    	sl_Exe_global<<<nBlocks, nThreads>>>(nloop, nBlocks, nThreads, maxProc, DmemSiz, IndexSiz, spReg,
								d_runParm, d_stk, d_GTbl, d_LTbl, d_Index, d_CodeArr, d_Dmem,
								d_gmem, d_nbrLITERAL, d_code, d_stack);
	}

    //hipMemcpy(h_Dmem, d_Dmem, sizeof(double)*DmemSiz * maxProc, hipMemcpyDeviceToHost);
    //hipMemcpy(h_Gmem, d_Gmem, sizeof(double)*GmemSiz, hipMemcpyDeviceToHost);
/*
	for (int i=0; i < GmemSiz; i++)
    {
        Gmem.set(i, h_Gmem[i]);
        //std::cout<<Gmem.get(i)<<endl;
	}
*/
	delete [] h_Index;
	hipFree(d_Index);
	delete [] h_CodeArr;
	hipFree(d_CodeArr);
	delete [] h_GTbl;
	delete [] h_LTbl;
	hipFree(d_GTbl);
	hipFree(d_LTbl);
	delete [] h_Dmem;
//	delete [] h_Gmem;
	delete [] h_nbrLITERAL;
	hipFree(d_Dmem);
//	hipFree(d_Gmem);
	hipFree(d_nbrLITERAL);

	hipFree(d_runParm);

	hipFree(d_stk);
	hipFree(d_stack);
	hipFree(d_code);
}

void sl_Print_h5(string& nm)
{
    int GSiz = Gtable.size();
    //std::string nm = name_;
    bool isin = false;

	for (int i=0; i < GSiz; i++)
    {
        string nm_g = Gtable[i].name;
        nm_g.erase(0,1);

		if ( nm_g == nm ) {
            isin = true;
			int aryLen = Gtable[i].aryLen;
			int adrs = Gtable[i].adrs;

			double* dList = new double[aryLen];

			for (int i2=0; i2 < aryLen; i2++) {
				dList[i2] = Gmem.get(adrs + i2);
			}
			H5Write( nm.c_str(), dList, aryLen);

			delete [] dList;
            break;
		}
	}

    if (isin == false) cout << "no file" << endl;
}


int InputDvar(char* name_, int aryLen_, double* Lists, IO io_)
{
	SymTbl sym;

	if (name_[0] == '$') {
		sym.name = name_;
	}
	else {
		string pre = "$";
		string p_nm = pre + name_;
		sym.name = p_nm;
	}
	sym.nmKind = devId;
	sym.dtTyp = DBL_T;
	sym.aryLen = aryLen_;
	sym.adrs = Gmem.size();
	sym.io = io_;

	Gtable.push_back(sym);

	int index = Gmem.mem.size();

	for (int i=0; i < aryLen_; i++) {
		Gmem.mem.push_back(Lists[i]);
	}

	return index;
}


void device_sl_exe(char fn[], int devId, int maxProc, double* host_List)
{
    hipSetDevice(devId);
    convert_to_internalCode(fn);
    syntaxChk();

    int  nBlocks, nThreads;
    nBlocks = 65535 ;
    
    hipDeviceSetLimit(hipLimitStackSize, 60*1024);
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, devId);

	nThreads = (int)(devProp.maxThreadsPerBlock / 2);

	sl_run_device(devId, maxProc, nBlocks, nThreads, host_List);
	
    intercode.resize(0);
	Ind.resize(0);
	Gtable.resize(0);
	Ltable.resize(0);
	nbrLITERAL.resize(0);
	Dmem.mem.resize(0);
}

void device_sl_syntax_check(char fn[])
{
	convert_to_internalCode(fn);
	syntaxChk();
}

void H5Write(const char* FILE, double* data, int NX)
{
	char* DATASETNAME = "data";
	int RANK = 1;

	hid_t       file, dataset;
    hid_t       datatype, dataspace;
    hsize_t     dimsf[RANK];
    herr_t      status;
    int         rows;
    int         i;


    rows = NX;

	string str1 = FILE;
	string str2 = str1 + ".h5";

    file = H5Fcreate(str2.c_str(), H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);

    dimsf[0] = NX;

    dataspace = H5Screate_simple(RANK, dimsf, NULL);

    datatype = H5Tcopy(H5T_NATIVE_DOUBLE);
    status = H5Tset_order(datatype, H5T_ORDER_LE);

    dataset = H5Dcreate(file, DATASETNAME, datatype, dataspace,
			H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT);

    status = H5Dwrite(dataset, H5T_NATIVE_DOUBLE, H5S_ALL, H5S_ALL,
		      H5P_DEFAULT, data);

    H5Sclose(dataspace);
    H5Tclose(datatype);
    H5Dclose(dataset);
    H5Fclose(file);
}

void H5Read(const char* FILE, double* data, int rows)
{
	char* DATASETNAME = "data";

    hid_t       file, dataset;
    hid_t       datatype, dataspace;
    H5T_class_t class_h5;
    H5T_order_t order;
    size_t      size;
    hsize_t     dims_out[1];
    herr_t      status;
    int          i, status_n, rank;

	string str = FILE;
	string str2 = str + ".h5";

    file = H5Fopen(str2.c_str(), H5F_ACC_RDONLY, H5P_DEFAULT);
    dataset = H5Dopen1(file, DATASETNAME);

    datatype  = H5Dget_type(dataset);
    class_h5     = H5Tget_class(datatype);
    if (class_h5 == H5T_INTEGER) printf("Data set has INTEGER type \n");
    order     = H5Tget_order(datatype);
    //if (order == H5T_ORDER_LE) printf("Little endian order \n");

    size  = H5Tget_size(datatype);

    dataspace = H5Dget_space(dataset);    // dataspace handle
    rank      = H5Sget_simple_extent_ndims(dataspace);
    status_n  = H5Sget_simple_extent_dims(dataspace, dims_out, NULL);

    rows = dims_out[0];

    status = H5Dread(dataset, H5T_NATIVE_DOUBLE, H5S_ALL, H5S_ALL, H5P_DEFAULT, data);

    // Close/release resources
    H5Tclose(datatype);
    H5Dclose(dataset);
    H5Sclose(dataspace);
    H5Fclose(file);
}

void InputDvarNoH5(char* name_, int aryLen_, IO io_)
{
	SymTbl sym;

	if (name_[0] == '$') {
		sym.name = name_;
	}
	else {
		string pre = "$";
		string p_nm = pre + name_;
		sym.name = p_nm;
	}
	sym.nmKind = devId;
	sym.dtTyp = DBL_T;
	sym.aryLen = aryLen_;
	sym.adrs = Gmem.size();
	sym.io = io_;

	Gtable.push_back(sym);

	double* dList = new double[aryLen_];

	for (int i=0; i < aryLen_; i++) {
		dList[i] = 0.;
		Gmem.mem.push_back(dList[i]);
	}
	string fn = sym.name;
	H5Write(name_, dList, aryLen_);

	delete [] dList;
}

void InputDvarYesH5(char* name_, int aryLen_, IO io_)
{
	SymTbl sym;
    string nm;

	if (name_[0] == '$') {
		nm = name_;
	}
	else {
		string pre = "$";
		string p_nm = pre + name_;
		nm = p_nm;
	}

    bool isin = false;

    int sizG = Gtable.size();
    for (int i=0; i < sizG; i++) {
        if (Gtable[i].name == nm) {

            isin = true;
            if (Gtable[i].aryLen == aryLen_) {
                double* dList = new double[aryLen_];
                string fn = Gtable[i].name ;
                H5Read(name_, dList, aryLen_);

                int adrs = Gtable[i].adrs;
                
                for (int i=0; i < aryLen_; i++) {
                    Gmem.mem[adrs + i] = dList[i];
                }

                delete [] dList;
            } 
            else {
                std::cout << "Not match array size" << std::endl;
            }

            
            break;
        }
    }

    if (isin == false) {

        sym.name = nm;
        sym.nmKind = devId;
        sym.dtTyp = DBL_T;
        sym.aryLen = aryLen_;
        sym.adrs = Gmem.size();
        sym.io = io_;

        Gtable.push_back(sym);

        double* dList = new double[aryLen_];
        string fn = sym.name ;
        H5Read(name_, dList, aryLen_);

        double add_step = 0.;
        for (int i=0; i < aryLen_; i++) {
            Gmem.mem.push_back(dList[i]);
        }

        delete [] dList;
    }
}

void loadcode(char fn[])
{
    /*if (access(fn), 0) != 0) {
        std::cout << "no " << fn << " file" << std::endl;
        return 0;
    }*/
    convert_to_internalCode(fn);
    syntaxChk();
}

void d_sl_exe(int devId, int maxProc)
{
    hipSetDevice(devId);
    //convert_to_internalCode(fn);
    //syntaxChk();

    int  nBlocks, nThreads;
    nBlocks = 65535 ;

    hipDeviceSetLimit(hipLimitStackSize, 60*1024);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, devId);

    nThreads = (int)(devProp.maxThreadsPerBlock / 2);

    sl_run_device_H5(devId, maxProc, nBlocks, nThreads);
}

void create_(char* name_, int NList, double *dList)
{
	SymTbl sym;
    string nm;

	if (name_[0] == '$') {
		nm = name_;
	}
	else {
		string pre = "$";
		string p_nm = pre + name_;
		nm = p_nm;
	}

    sym.name = nm;
    sym.nmKind = devId;
    sym.dtTyp = DBL_T;
    sym.aryLen = NList;
    sym.adrs = Gmem.size();
    sym.io = Out;

    Gtable.push_back(sym);

    for (int i=0; i < NList; i++) {
        Gmem.mem.push_back(dList[i]);
    }
}

void update_(char* name_, int NList, double *dList)
{
    if (d_gmem != NULL) {
        cpymemDeviceToHost();
        hipFree(d_gmem);
    }

    SymTbl sym;
    string nm;

    if (name_[0] == '$') {
        nm = name_;
    }
    else {
        string pre = "$";
        string p_nm = pre + name_;
        nm = p_nm;
    }

    int sizG = Gtable.size();
    for (int i=0; i < sizG; i++) {
        if (Gtable[i].name == nm) {

            if (Gtable[i].aryLen == NList) {

                int adrs = Gtable[i].adrs;

                for (int i=0; i < NList; i++) {
                    Gmem.mem[adrs + i] = dList[i];
                }

            }
            else {
                std::cout << "Not match array size" << std::endl;
            }

            break;
        }
    }

    cpymemHostToDevice();

}

int get_length(char* name_)
{
    SymTbl sym;
    string nm;

    if (name_[0] == '$') {
        nm = name_;
    }
    else {
        string pre = "$";
        string p_nm = pre + name_;
        nm = p_nm;
    }

    int size = 0;

    int sizG = Gtable.size();
    for (int i=0; i < sizG; i++) {
        if (Gtable[i].name == nm) {

            size = Gtable[i].aryLen;

            break;
        }
    }
    return size;
}

void get_(char* name_, double *dList)
{
//    cpymemDeviceToHost();

    SymTbl sym;
    string nm;

    if (name_[0] == '$') {
        nm = name_;
    }
    else {
        string pre = "$";
        string p_nm = pre + name_;
        nm = p_nm;
    }

    int sizG = Gtable.size();
    for (int i=0; i < sizG; i++) {

        if (Gtable[i].name == nm) {
            int adrs = Gtable[i].adrs;
            int NList = Gtable[i].aryLen;

            for (int i=0; i < NList; i++) {
                dList[i] = Gmem.mem[adrs + i];
            }

            break;
        }
    }
}

void cpymemHostToDevice()
{
    int gSiz = Gmem.mem.size();

    double* h_Gmem = new double[gSiz];

    for (int i=0; i< gSiz; i++) h_Gmem[i] = Gmem.get(i);

    if (hipSuccess != hipMalloc(&d_gmem, sizeof(double)*gSiz))
    {
        std::cout << "Memory Over" << std::endl;
        return;
    }
    hipMemcpy(d_gmem, h_Gmem, sizeof(double)*gSiz, hipMemcpyHostToDevice);

    delete [] h_Gmem;
}

void cpymemDeviceToHost()
{
    int gSiz = Gmem.mem.size();

    double* h_Gmem = new double[gSiz];

    hipMemcpy(h_Gmem, d_gmem, sizeof(double)*gSiz, hipMemcpyDeviceToHost);

    for (int i=0; i < gSiz; i++) Gmem.set(i, h_Gmem[i]);

    delete [] h_Gmem;
}

void end_()
{
    hipFree(d_gmem);

    intercode.clear();
    Ind.clear();
    Gtable.clear();
    Ltable.clear();
    nbrLITERAL.clear();
    strLITERAL.clear();
    Gmem.mem.clear();
    Dmem.mem.clear();
}
