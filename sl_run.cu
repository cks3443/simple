#include "hip/hip_runtime.h"
#include "sl_run.cuh"

__global__
void sl_Exe_global(int nloop, int nBlocks, int nThreads, unsigned int maxProc, int DmemSiz, int IndexSiz, int CodeArrSiz, int spReg,
     RUN_PARM* x_runParm, Stack* x_stk, d_SymTbl* GTbl, d_SymTbl* LTbl,
     int* x_Index, int* x_CodeArr, double* x_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* x_code, double* x_stack)
{
    unsigned int lo_id = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int thread_id = (unsigned int)nBlocks * (unsigned int)nThreads * (unsigned int)nloop + lo_id;
    
    if (thread_id < maxProc) {
        int* Index = &(x_Index[thread_id * IndexSiz]);
        int* CodeArr = &(x_CodeArr[thread_id * CodeArrSiz]);
        RUN_PARM* d_runParm = &(x_runParm[thread_id]);
        Stack* d_stk = &(x_stk[thread_id]);
        
        double* d_Dmem = &(x_Dmem[thread_id * DmemSiz]);
        
        TokenSet* d_code = &(x_code[2*thread_id]);
        double* d_stack = &(x_stack[MAXSIZE_ * thread_id]);

        d_stk->MAXSIZE = MAXSIZE_;
    	d_stk->top = -1;

        d_runParm->baseReg = 0;
        d_runParm->spReg = spReg;
        d_runParm->Pc = 1;

        d_runParm->ThreadId = thread_id;

        d_runParm->break_Flg=d_runParm->return_Flg=d_runParm->exit_Flg=false;
        d_runParm->maxLine = IndexSiz-2;

        sl_execute(d_runParm, d_stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, d_code, d_stack);
    }
}

__device__ __host__
void sl_execute(RUN_PARM* runParm, Stack* stk, d_SymTbl* GTbl, d_SymTbl* LTbl,
                int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack)
{
    while (runParm->Pc <= runParm->maxLine && ! runParm->exit_Flg) {
        sl_statement(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
    }
}

__device__ __host__
void sl_statement(RUN_PARM* runParm, Stack* stk, d_SymTbl* GTbl, d_SymTbl* LTbl,
                int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack)
{
    TokenSet* save = &(code[1]);
    int top_line, end_line, varAdrs;
    double wkVal, endDt, stepDt;

    if (runParm->Pc > runParm->maxLine || runParm->exit_Flg) return;

    sl_firstCode(code, runParm, runParm->Pc, Index, CodeArr, nbrLITERAL);

    top_line = runParm->Pc;
    end_line = code->jmpAdrs;

    if (code->kind == If) end_line = sl_endline_of_If(runParm, save, runParm->Pc, Index, CodeArr, nbrLITERAL);

    code[1]=code[0];

    if (code->kind == If) {
        if (sl_get_expression(runParm, stk, While, EofLine, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack)) {
            ++runParm->Pc;
			sl_block(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
            runParm->Pc = end_line + 1;
            return ;
        }
        runParm->Pc = save->jmpAdrs;
        while (sl_lookCode(runParm->Pc, Index, CodeArr) == Elif)
        {
			sl_firstCode(save, runParm, runParm->Pc, Index, CodeArr, nbrLITERAL);
		    sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
            sl_expression(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
            
            if (stack_pop(stk, stack)) {
                ++runParm->Pc;
                sl_block(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
                runParm->Pc = end_line + 1;
                return ;
            }
            runParm->Pc = save->jmpAdrs;
        }

        if (sl_lookCode(runParm->Pc, Index, CodeArr) == Else) {
            ++runParm->Pc;
            sl_block(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
            runParm->Pc = end_line + 1;
            return ;
        }
        ++runParm->Pc;
	}
	else if (code->kind == While) {

		for (;;) {
			if (!sl_get_expression(runParm, stk, While, EofLine, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack) ) break;
			++runParm->Pc;
			sl_block(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);

			if (runParm->break_Flg || runParm->return_Flg || runParm->exit_Flg) {
				runParm->break_Flg = false;
				break;
			}
			runParm->Pc = top_line;
			sl_firstCode(code, runParm, runParm->Pc, Index, CodeArr, nbrLITERAL);
		}
		runParm->Pc = end_line + 1;
	}
	else if (code->kind == For) {
		sl_nextCode(save, runParm, nbrLITERAL, Index, CodeArr);
		varAdrs = sl_get_memAdrs(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, save, stack);

		sl_expression(runParm, stk, '=', 0, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		sl_set_dtTyp(runParm, stk, save, DBL_T, GTbl, LTbl, d_Dmem, d_Gmem);

		sl_set_mem(d_Dmem, varAdrs, stack_pop(stk, stack));

		endDt = sl_get_expression(runParm, stk, To, 0, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);

		if (code->kind == Step) {
			stepDt = sl_get_expression(runParm, stk, Step, 0, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		}
		else {
			stepDt = 1.;
		}

		for (;; runParm->Pc = top_line)
		{                            
			if (stepDt >= 0) {                                
				if (d_Dmem[varAdrs] > endDt) break;            
			}
			else {                                        
				if (d_Dmem[varAdrs] < endDt) break;
			}                                                 
			++runParm->Pc;
			sl_block(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);

			if (runParm->break_Flg || runParm->return_Flg || runParm->exit_Flg) {
				runParm->break_Flg = false;
				break;                       
			}
			d_Dmem[varAdrs] += stepDt;
		}                                                 
		runParm->Pc = end_line + 1; 
    }
    else if (code->kind == Break) {
    	runParm->break_Flg = true;
    }
    else if (code->kind == Gvar || code->kind == Lvar || code->kind == Dvar) {
		varAdrs = sl_get_memAdrs(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		
		int Op = 0;
		if (code->kind == SumAssign) Op = 1;
		else if (code->kind == MinusAssign) Op = 2;
		else if (code->kind == MultiAssign) Op = 3;
		else if (code->kind == DiviAssign) Op = 4;

		if (Op == 0) sl_expression(runParm, stk, '=', 0, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		else if (Op == 1) sl_expression(runParm, stk, SumAssign, 0, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		else if (Op == 2) sl_expression(runParm, stk, MinusAssign, 0, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		else if (Op == 3) sl_expression(runParm, stk, MultiAssign, 0, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		else if (Op == 4) sl_expression(runParm, stk, DiviAssign, 0, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);

		double Val;
		if (stk->top != -1) {
			Val = stack[stk->top];
			stk->top -= 1;
		}
		else Val = -12133131;


		sl_set_dtTyp(runParm, stk, save, DBL_T, GTbl, LTbl, d_Dmem, d_Gmem);

		if (Op == 0) {
			if (save->kind == Dvar) sl_set_mem(d_Gmem, varAdrs, Val);
			else if (save->kind == Gvar) sl_set_mem(d_Dmem, varAdrs, Val);
			else sl_set_mem(d_Dmem, varAdrs, Val);
		}
		else if (Op == 1) {
			if (save->kind == Dvar) sl_add_mem(d_Gmem, varAdrs, Val);
			else if (save->kind == Gvar) sl_add_mem(d_Dmem, varAdrs, Val);
			else sl_add_mem(d_Dmem, varAdrs, Val);
		}
		else if (Op == 2) {
			if (save->kind == Dvar) d_Gmem[varAdrs] -= Val;
			else if (save->kind == Gvar) d_Dmem[varAdrs] -= Val;
			else d_Dmem[varAdrs] -= Val;
		}
		else if (Op == 3) {
			if (save->kind == Dvar) d_Gmem[varAdrs] *= Val;
			else if (save->kind == Gvar) d_Dmem[varAdrs] *= Val;
			else d_Dmem[varAdrs] *= Val;
		}
		else if (Op == 4) {
			if (save->kind == Dvar) d_Gmem[varAdrs] /= Val;
			else if (save->kind == Gvar) d_Dmem[varAdrs] /= Val;
			else d_Dmem[varAdrs] /= Val;
		}

        ++ runParm->Pc;
    }
    else if (code->kind == Option || code->kind == Var || code->kind == EofLine) {
    	++ runParm->Pc;
    }
}

__device__ __host__
void sl_firstCode(TokenSet* code, RUN_PARM* runParm, int line,
                    int* Index, int* CodeArr, double* nbrLITERAL)
{
    runParm->code_ptr = Index[line];

    TknKind k = (TknKind)CodeArr[runParm->code_ptr];

    if (k==If || k == For || k == Elif || k == Else || k == End || k == While) {
    	runParm->code_ptr++;
    	int jmpAdrs = CodeArr[runParm->code_ptr++];
        token_set(code, k, -1, jmpAdrs);
    }
    else sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
}

__device__ __host__
void sl_nextCode(TokenSet* Ts, RUN_PARM* runParm, double* nbrLITERAL, int* Index, int* CodeArr)
{
    TknKind kd;
    int jmpAdrs, tblNbr, nK;

    if ((TknKind)CodeArr[runParm->code_ptr] == EofLine) {
        token_set(Ts, EofLine);
    }

    nK = CodeArr[runParm->code_ptr];
    kd = (TknKind)CodeArr[runParm->code_ptr++];

    switch(kd) 
	{
    case IntNum: case DblNum:
        tblNbr = CodeArr[runParm->code_ptr++];
        Ts->kind = kd;
        Ts->nKind = nK;
        Ts->dblVal = nbrLITERAL[tblNbr];
        break;

    case Gvar: case Lvar: case Dvar:
        tblNbr = CodeArr[runParm->code_ptr++];
        Ts->kind=kd;
        Ts->nKind = nK;
        Ts->symNbr = tblNbr;
        Ts->dblVal = -1;
        break;

    default:
    	Ts->kind = kd;
        Ts->nKind = nK;
        break;
    }

}

__device__ __host__
int sl_endline_of_If(RUN_PARM* runParm, TokenSet* cd, int line, int* Index, int* CodeArr, double* nbrLITERAL)
{
    int jmpline;
    int save_code_ptr = runParm->code_ptr;
    int save_Pc = runParm-> Pc;

    sl_firstCode(cd, runParm, line, Index, CodeArr, nbrLITERAL);

    for (;;) {
        jmpline = cd->jmpAdrs;
        sl_firstCode(cd, runParm, jmpline, Index, CodeArr, nbrLITERAL);
        if (cd->kind == Elif || cd->kind == Else) continue;
        if (cd->kind == End) break;
    }
	runParm->code_ptr = save_code_ptr;
	runParm->Pc = save_Pc;

    return jmpline;
}

__device__ __host__
double sl_get_expression(RUN_PARM* runParm, Stack* stk, int kind1, int kind2, d_SymTbl* GTbl, d_SymTbl* LTbl,
                    int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack)
{
    sl_expression(runParm, stk, kind1, kind2, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
    return stack_pop(stk, stack);
}


__device__ __host__
void sl_expression(RUN_PARM* runParm, Stack* stk, int kind1, int kind2, d_SymTbl* GTbl, d_SymTbl* LTbl,
                    int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack)
{
    if (kind1 != 0) sl_chk_nextCode(runParm, code, kind1, nbrLITERAL, Index, CodeArr);
    sl_expression(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
    if (kind2 !=0) sl_chk_nextCode(runParm, code, kind2, nbrLITERAL, Index, CodeArr);
}


__device__ __host__
void sl_chk_nextCode(RUN_PARM* runParm, TokenSet* code, int kind2, double* nbrLITERAL,
                            int* Index, int* CodeArr)
{
    sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
}


__device__ __host__
void sl_expression(RUN_PARM* runParm, Stack* stk, d_SymTbl* GTbl, d_SymTbl* LTbl,
            int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack)
{
    TknKind op;
    int nK;
    sl_term(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack, 1);

    while(true) {
        if (code->nKind != 43 && code->nKind != 45) break;

        nK = code->nKind;
        sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
        sl_term(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack, 1);
        sl_binaryExpr(stk, op, stack);
    }
}

__device__ __host__
void sl_term(RUN_PARM* runParm, Stack* stk, d_SymTbl* GTbl, d_SymTbl* LTbl,
            int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack,
            int n)
{
    TknKind op;

    sl_factor(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
        
    int nK = code->nKind;
    while (nK == 47 || nK == 42 || nK == 92 ||
        nK == 37 || nK == 166 || nK == 167 ||
        nK == 168 || nK == 169 || nK == 170 ||
        nK == 171 || nK == 172 || nK == 173)
    {
        sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
        sl_factor(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem,nbrLITERAL, code, stack);
        sl_binaryN(stk, nK, stack);
        nK = code->nKind;
    }
/*
    int nK = code->nKind;

	if (n == 7) {
        sl_factor(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
        return;
    }
	
    sl_term(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack, n+1);
    
	while (n == sl_opOrder(code->nKind))
	{
        nK = code->nKind;
		sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
        sl_term(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack, n+1);
		sl_binaryN(stk, nK, stack);
    }
*/
}

__device__ __host__
void sl_factor(RUN_PARM* runParm, Stack* stk, d_SymTbl* GTbl, d_SymTbl* LTbl,
                int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack)
{
	TknKind kd, k;
	kd = k = code->kind;

	int adr=0, index, len, symNbr, Adrs=0, symNbr_2, adr_2;
	double d=0.;

    switch (kd) {
		
	case EXP:
		sl_expression(runParm, stk, Lparen, Rparen, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
        stack_push(stk, exp(stack_pop(stk, stack)), stack);
        break;

	case LOG:
		sl_expression(runParm, stk, Lparen, Rparen, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
        stack_push(stk, log(stack_pop(stk, stack)), stack);
        break;

	case PID:
		stk->top += 1;
		stack[stk->top] = (double)runParm->ThreadId;
		sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
		break;
	
	case Not: case Minus: case Plus:
		sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
		sl_factor(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		if (kd == Not) stack_push(stk, !(stack_pop(stk, stack)), stack);
		if (kd == Minus) stack_push(stk, -(stack_pop(stk, stack)), stack);
		break;

	case Lparen:
		sl_expression(runParm, stk, Lparen, Rparen, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
		break;

    case IntNum: case DblNum:
        stack_push(stk, code->dblVal, stack);
        sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
        break;

    case Gvar: case Lvar: case Dvar:

    	sl_chk_dtTyp(code, GTbl, LTbl);

		symNbr = code->symNbr;

		d_SymTbl* sym;

		if (code->kind == Dvar) sym = &(GTbl[symNbr]);
		else if (code->kind == Lvar) sym = &(LTbl[symNbr]);
		else sym = &(GTbl[symNbr]);

		adr = sl_get_topAdrs(runParm, code, symNbr, GTbl, LTbl);
		len = sym->aryLen;

		if (Lbracket == (TknKind) CodeArr[runParm->code_ptr]) {
			sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
			sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);

			if (code->kind == Lvar || code->kind == Gvar) {
				int symNbr_2 = code->symNbr;
				int adr_2 = sl_get_topAdrs(runParm, code, symNbr_2, GTbl, LTbl);
				d = sl_get_mem(d_Dmem, adr_2);
			}
			else if (code->kind == IntNum || code->kind == DblNum) {
				d = code->dblVal;
			}
			else if (code->kind == PID) {
				d = (double)runParm->ThreadId;
			}

			sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
		}

		index = (int)d;

		if (len==0)	Adrs = adr;
		else		Adrs = adr+index;

        if (kd == Gvar) stack_push(stk, sl_get_mem(d_Dmem, Adrs), stack);
        else if (kd == Lvar) stack_push(stk, sl_get_mem(d_Dmem, Adrs), stack);
        else if (kd == Dvar) stack_push(stk, sl_get_mem(d_Gmem, Adrs), stack);

        sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);

        break;
    }

}


__device__ __host__
void sl_chk_dtTyp(const TokenSet* cd, d_SymTbl* GTbl, d_SymTbl* LTbl)
{
    int symNbr = cd->symNbr;

    if (cd->kind == Dvar) {
        if (GTbl[symNbr].dtTyp == NON_T) {
        	return;
        }
    }
    else if (cd->kind == Lvar) {
        if (LTbl[symNbr].dtTyp == NON_T) {
        	return;
        }
    }
	else if (cd->kind == Gvar)
	{
		if (GTbl[symNbr].dtTyp == NON_T)
		{
			return;
		}
	}
}


__device__ __host__
int sl_get_memAdrs(RUN_PARM* runParm, Stack* stk, d_SymTbl* GTbl, d_SymTbl* LTbl,
        int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack)
{
    int adr=0, index, len, symNbr;
    symNbr = code->symNbr;
    double d=0.;

    d_SymTbl* sym;


    if (code->kind == Dvar) sym = &(GTbl[symNbr]);
    else if (code->kind == Lvar) sym = &(LTbl[symNbr]);
    else sym = &(GTbl[symNbr]);

    adr = sl_get_topAdrs(runParm, code, symNbr, GTbl, LTbl);
    len = sym->aryLen;
    sl_nextCode(code, runParm, nbrLITERAL, Index, CodeArr);
    if (len==0) return adr;

    d = sl_get_expression(runParm, stk, '[',']', GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);

    index = (int)d;

    return adr+index;
}


__device__ __host__
int sl_get_topAdrs(RUN_PARM* runParm, const TokenSet* cd, int symNbr, d_SymTbl* GTbl, d_SymTbl* LTbl)
{
    switch(cd->kind) {
    case Gvar: case Dvar:
        return GTbl[symNbr].adrs;
    case Lvar:
        return LTbl[symNbr].adrs + runParm->baseReg;
    }
}


__device__ __host__
void sl_set_dtTyp(RUN_PARM* runParm, Stack* stk, TokenSet* cd, DtType typ, d_SymTbl* GTbl, d_SymTbl* LTbl,
                    double* d_Dmem, double* d_Gmem)
{
    int memAdrs = sl_get_topAdrs(runParm, cd, cd->symNbr, GTbl, LTbl);

    d_SymTbl* sym;


    if (cd->kind == Dvar) sym = &(GTbl[cd->symNbr]);
    else if (cd->kind == Lvar) sym = &(LTbl[cd->symNbr]);
    else sym = &(GTbl[cd->symNbr]);

    if (sym->dtTyp != NON_T) return;

    sym->dtTyp = typ;
}


__device__ __host__
void sl_block(RUN_PARM* runParm, Stack* stk, d_SymTbl* GTbl, d_SymTbl* LTbl,
        int* Index, int* CodeArr, double* d_Dmem, double* d_Gmem, double* nbrLITERAL, TokenSet* code, double* stack)
{
    TknKind k;
    while (! runParm->break_Flg && ! runParm->return_Flg && ! runParm->exit_Flg) {
        k = sl_lookCode(runParm->Pc, Index, CodeArr);
        if (k == Elif || k == Else || k == End) break;
        sl_statement(runParm, stk, GTbl, LTbl, Index, CodeArr, d_Dmem, d_Gmem, nbrLITERAL, code, stack);
    }
}


__device__ __host__
TknKind sl_lookCode( int line, int* Index, int* CodeArr)
{
	int Posi = Index[line];
    return (TknKind)CodeArr[Posi];
}

__device__ __host__
void sl_binaryExpr(Stack* stk, TknKind op, double* stack)
{
	double d2 = stack_pop(stk, stack), d1 = stack_pop(stk, stack);

	if ((op==Divi || op==Mod || op==IntDivi) && d2==0) {
		return;
	}

	switch (op) {
	case Plus:    stack_push(stk, d1 + d2, stack);  break;
	case Minus:   stack_push(stk, d1 - d2, stack);  break;
	case Multi:   stack_push(stk, d1 * d2, stack);  break;
	case Divi:    stack_push(stk, d1 / d2, stack);  break;
	case Mod:     stack_push(stk, (int)d1 % (int)d2, stack); break;
	case IntDivi: stack_push(stk, (int)d1 / (int)d2, stack); break;
	case Less:    stack_push(stk, d1 <  d2, stack); break;
	case LessEq:    stack_push(stk, d1 <=  d2, stack); break;
	case Great:    stack_push(stk, d1 >  d2, stack); break;
	case GreatEq:    stack_push(stk, d1 >=  d2, stack); break;
	case Equal:    stack_push(stk, d1 == d2, stack); break;
	case NotEq:    stack_push(stk, d1 != d2, stack); break;
	case And:    stack_push(stk, d1 && d2, stack); break;
	case Or:    stack_push(stk, d1 || d2, stack); break;

	}
}

__device__ __host__
void sl_binaryN(Stack* stk, int op, double* stack)
{
	double d2 = stack_pop(stk, stack), d1 = stack_pop(stk, stack);

	if ((op== 47 || op==37 || op==92) && d2==0) {
		return;
	}

	if (op == 43)    stack_push(stk, d1 + d2, stack);
	else if (op ==45) stack_push(stk, d1 - d2, stack);
	else if (op ==42) stack_push(stk, d1 * d2, stack);
	else if (op ==47) stack_push(stk, d1 / d2, stack);
	else if (op ==37) stack_push(stk, (int)d1 % (int)d2, stack);
	else if (op ==92) stack_push(stk, (int)d1 / (int)d2, stack);
	else if (op ==168) stack_push(stk, d1 <  d2, stack);
	else if (op ==169) stack_push(stk, d1 <=  d2, stack);
	else if (op ==170) stack_push(stk, d1 >  d2, stack);
	else if (op ==171) stack_push(stk, d1 >=  d2, stack);
	else if (op ==166) stack_push(stk, d1 == d2, stack);
	else if (op ==167) stack_push(stk, d1 !=  d2, stack);
	else if (op ==172) stack_push(stk, d1 && d2, stack);
	else if (op ==173) stack_push(stk, d1 ||  d2, stack);
}

__device__ __host__
int sl_opOrder(int nK)
{
    switch (nK) {
    case 42: case 47: case 37:
    case 92:                    return 6;
    case 43:  case 45:          return 5;
    case 168:  case 169:
    case 170: case 171:        return 4;
    case 166: case 167:          return 3;
    case 172:                        return 2;
    case 173:                         return 1;
    default:                         return 0;
    }
}

__device__ __host__
void sl_set_mem(double* mem, int adrs, double dt)
{
    mem[adrs] = dt;
}

__device__ __host__
void sl_add_mem(double* mem, int adrs, double dt)
{
   mem[adrs] += dt;
}

__device__ __host__
double sl_get_mem(double* mem, int adrs)
{
    return mem[adrs];
}

__device__ __host__
void token_clear(TokenSet* Ts)
{
    Ts->kind=Others;
    Ts->dblVal=0.;
    Ts->symNbr=0;
    Ts->jmpAdrs=0;
}

__device__ __host__
void token_set(TokenSet* Ts)
{
    token_clear(Ts);
}

__device__ __host__
void token_set(TokenSet* Ts, TknKind k)
{
    token_clear(Ts);
    Ts->kind=k;
}


__device__ __host__
void token_set(TokenSet* Ts, TknKind k, double d)
{
    token_clear(Ts);
    Ts->kind=k;
    Ts->dblVal=d;
}


__device__ __host__
void token_set(TokenSet* Ts, TknKind k, int sym, int jmp)
{
    token_clear(Ts);
    Ts->kind=k;
    Ts->symNbr=sym;
    Ts->jmpAdrs=jmp;
}


__device__ __host__
bool stack_isfull(Stack* St)
{
    if (St->top == St->MAXSIZE) return true;
    else return false;
}

__device__ __host__
bool stack_empty(Stack* St)
{
    if (St->top == -1) return true;
    else  return false;
}

__device__ __host__
void stack_push(Stack* St, double data, double* stack)
{
    if (!stack_isfull(St)) {
        St->top = St->top + 1;
        stack[St->top]=data;
    }
}

__device__ __host__
double stack_pop(Stack* St, double* stack)
{
    double data;

    if (St->top != -1) {
        data = stack[St->top];
        St->top -= 1;
        return data;
    }
}
